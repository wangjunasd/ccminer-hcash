#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include <cuda_helper.h>
#include <miner.h>


#define  F(x, y, z) (((x) ^ (y) ^ (z)))
#define FF(x, y, z) (((x) & (y)) | ((x) & (z)) | ((y) & (z)))
#define GG(x, y, z) ((z)  ^ ((x) & ((y) ^ (z))))

#define P0(x) x ^ ROTL32(x,  9) ^ ROTL32(x, 17)
#define P1(x) x ^ ROTL32(x, 15) ^ ROTL32(x, 23)

__device__
void sm3_compress1(uint32_t digest[8], const uint32_t block[16]){
	uint32_t tt1, tt2, i, ss1, ss2, x, y;
	uint32_t w[68];
	uint32_t a = digest[0];
	uint32_t b = digest[1];
	uint32_t c = digest[2];
	uint32_t d = digest[3];
	uint32_t e = digest[4];
	uint32_t f = digest[5];
	uint32_t g = digest[6];
	uint32_t h = digest[7];

#pragma unroll 16
	for (i = 0; i<16; i++) {
		w[i] = cuda_swab32(block[i]);
	}
#pragma unroll 52
	for (i = 16; i<68; i++) {
		x = ROTL32(w[i - 3], 15);
		y = ROTL32(w[i - 13], 7);

		x ^= w[i - 16];
		x ^= w[i - 9];
		y ^= w[i - 6];

		w[i] = P1(x) ^ y;
	}
#pragma unroll 16
	for (i = 0; i < 16; i++){

		ss2 = ROTL32(a, 12);
		ss1 = ROTL32(ss2 + e + ROTL32(0x79cc4519, i), 7);
		ss2 ^= ss1;

		tt1 = d + ss2 + (w[i] ^ w[i + 4]) + F(a, b, c);
		tt2 = h + ss1 + w[i] + F(e, f, g);

		d = c;
		c = ROTL32(b, 9);
		b = a;
		a = tt1;
		h = g;
		g = ROTL32(f, 19);
		f = e;
		e = P0(tt2);

	}
#pragma unroll 48
	for (i = 16; i < 64; i++){
		ss2 = ROTL32(a, 12);
		ss1 = ROTL32(ss2 + e + ROTL32(0x7a879d8a, i), 7);
		ss2 ^= ss1;

		tt1 = d + ss2 + (w[i] ^ w[i + 4]) + FF(a, b, c);
		tt2 = h + ss1 + w[i] + GG(e, f, g);

		d = c;
		c = ROTL32(b, 9);
		b = a;
		a = tt1;
		h = g;
		g = ROTL32(f, 19);
		f = e;
		e = P0(tt2);

	}

	digest[0] ^= a;
	digest[1] ^= b;
	digest[2] ^= c;
	digest[3] ^= d;
	digest[4] ^= e;
	digest[5] ^= f;
	digest[6] ^= g;
	digest[7] ^= h;
}

__device__
void sm3_compress2(uint32_t digest[8]){
	uint32_t tt1, tt2, i, ss1, ss2;
	uint32_t w[68] = { 0x80000000, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x0, 0x200, 0x80404000, 0x0, 0x1008080, 0x10005000, 0x0, 0x2002a0, 0xac545c04, 0x0, 0x9582a39, 0xa0003000, 0x0, 0x200280, 0xa4515804, 0x20200040, 0x51609838, 0x30005701, 0xa0002000, 0x8200aa, 0x6ad525d0, 0xa0e0216, 0xb0f52042, 0xfa7073b0, 0x20000000, 0x8200a8, 0x7a542590, 0x22a20044, 0xd5d6ebd2, 0x82005771, 0x8a202240, 0xb42826aa, 0xeaf84e59, 0x4898eaf9, 0x8207283d, 0xee6775fa, 0xa3e0e0a0, 0x8828488a, 0x23b45a5d, 0x628a22c4, 0x8d6d0615, 0x38300a7e, 0xe96260e5, 0x2b60c020, 0x502ed531, 0x9e878cb9, 0x218c38f8, 0xdcae3cb7, 0x2a3e0e0a, 0xe9e0c461, 0x8c3e3831, 0x44aaa228, 0xdc60a38b, 0x518300f7 };
	uint32_t a = digest[0];
	uint32_t b = digest[1];
	uint32_t c = digest[2];
	uint32_t d = digest[3];
	uint32_t e = digest[4];
	uint32_t f = digest[5];
	uint32_t g = digest[6];
	uint32_t h = digest[7];

#pragma unroll 16
	for (i = 0; i < 16; i++){

		ss2 = ROTL32(a, 12);
		ss1 = ROTL32(ss2 + e + ROTL32(0x79cc4519, i), 7);
		ss2 ^= ss1;

		tt1 = d + ss2 + (w[i] ^ w[i + 4]) + F(a, b, c);
		tt2 = h + ss1 + w[i] + F(e, f, g);

		d = c;
		c = ROTL32(b, 9);
		b = a;
		a = tt1;
		h = g;
		g = ROTL32(f, 19);
		f = e;
		e = P0(tt2);

	}
#pragma unroll 48
	for (i = 16; i < 64;i++){
		ss2 = ROTL32(a, 12);
		ss1 = ROTL32(ss2 + e + ROTL32(0x7a879d8a, i), 7);
		ss2 ^= ss1;

		tt1 = d + ss2 + (w[i] ^ w[i + 4]) + FF(a, b, c);
		tt2 = h + ss1 + w[i] + GG(e, f, g);

		d = c;
		c = ROTL32(b, 9);
		b = a;
		a = tt1;
		h = g;
		g = ROTL32(f, 19);
		f = e;
		e = P0(tt2);

	}

	digest[0] ^= a;
	digest[1] ^= b;
	digest[2] ^= c;
	digest[3] ^= d;
	digest[4] ^= e;
	digest[5] ^= f;
	digest[6] ^= g;
	digest[7] ^= h;

}


/***************************************************/
// GPU Hash Function
__global__ void x14_sm3_gpu_hash_64(uint32_t threads, uint32_t *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		

		uint32_t *Hash = &g_hash[thread << 4];

		//__syncthreads();

		uint32_t digest[8] = {
			0x7380166F, 0x4914B2B9, 0x172442D7, 0xDA8A0600, 0xA96F30BC, 0x163138AA, 0xE38DEE4D, 0xB0FB0E4E
		};

		sm3_compress1(digest, Hash);

		sm3_compress2(digest);

#pragma unroll 8
		for (int i = 0; i < 8; i++)
			Hash[i] = cuda_swab32(digest[i]);
//#pragma unroll 8
//		for (int i = 8; i < 16; i++)
//			Hash[i] = 0;

	}

}

__host__ void x14_sm3_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash)
{

	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	x14_sm3_gpu_hash_64 << <grid, block >> >(threads, d_hash);
}